#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h> // Required for fabs()

#define RANGE 17.78
#define BLOCK_NUM 8         // Default grid size for final submission
#define THREADS_NUM 500     // Default block size for final submission

/*** TODO: insert the declaration of the kernel function below this line ***/
__global__ void vecGPU(float *ad, float *bd, float *cd, int n);
/**** end of the kernel declaration ***/


int main(int argc, char *argv[]){

	int n = 0; //number of elements in the arrays
	int i;  //loop index
	float *a, *b, *c; // The arrays that will be processed in the host.
	float *temp;  //array in host used in the sequential code.
	float *ad, *bd, *cd; //The arrays that will be processed in the device.
	clock_t start, end; // to meaure the time taken by a specific part of code
	
	if(argc != 2){
		printf("usage:  ./vectorprog n\n");
		printf("n = number of elements in each vector\n");
		exit(1);
		}
		
	n = atoi(argv[1]);
	printf("Each vector will have %d elements\n", n);
	
	
	//Allocating the arrays in the host
	size_t size = n * sizeof(float);
	
	if( !(a = (float *)malloc(size)) )
	{
	   printf("Error allocating array a\n");
	   exit(1);
	}
	
	if( !(b = (float *)malloc(size)) )
	{
	   printf("Error allocating array b\n");
	   exit(1);
	}
	
	if( !(c = (float *)malloc(size)) )
	{
	   printf("Error allocating array c\n");
	   exit(1);
	}
	
	if( !(temp = (float *)malloc(size)) )
	{
	   printf("Error allocating array temp\n");
	   exit(1);
	}
	
    //Fill out the arrays with random numbers between 0 and RANGE;
    srand((unsigned int)time(NULL));
    for (i = 0; i < n;  i++){
        a[i] = ( (float) rand() / (float) (RAND_MAX) ) * RANGE;
        b[i] = ( (float) rand() / (float) (RAND_MAX) ) * RANGE;
        c[i] = ( (float) rand() / (float) (RAND_MAX) ) * RANGE;
        temp[i] = c[i]; //temp is just another copy of C
    }
    
    //The sequential part
	start = clock();
	for(i = 0; i < n; i++)
		temp[i] += a[i] * b[i];
	end = clock();
	printf("Total time taken by the sequential part = %lf seconds\n", (double)(end - start) / CLOCKS_PER_SEC);

    /****************** The start GPU part: Do not modify anything in main() above this line  ************/
	//The GPU part
	
    // Create CUDA events for accurate timing
    hipEvent_t gpu_start, gpu_stop;
    float gpu_elapsed_time;
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);

	/* TODO: in this part you need to do the following:
		1. allocate ad, bd, and cd in the device
		2. send a, b, and c to the device  
	*/
    hipMalloc((void **)&ad, size);
    hipMalloc((void **)&bd, size);
    hipMalloc((void **)&cd, size);

    // Record the start event
    hipEventRecord(gpu_start);

    hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
    hipMemcpy(bd, b, size, hipMemcpyHostToDevice);
    hipMemcpy(cd, c, size, hipMemcpyHostToDevice);
		
	/* TODO: 	
		3. write the kernel, call it: vecGPU
		4. call the kernel (the kernel itself will be written at the comment at the end of this file), 
		   you need to decide about the number of threads, blocks, etc and their geometry.
	*/
    vecGPU<<<BLOCK_NUM, THREADS_NUM>>>(ad, bd, cd, n);
		
	/* TODO: 
		5. bring the cd array back from the device and store it in c array (declared earlier in main)
		6. free ad, bd, and cd
	*/
	hipMemcpy(c, cd, size, hipMemcpyDeviceToHost);

    // Record the stop event and synchronize
    hipEventRecord(gpu_stop);
    hipEventSynchronize(gpu_stop);

    // Calculate the elapsed time
    hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);

    hipFree(ad);
    hipFree(bd);
    hipFree(cd);
	
	printf("Total time taken by the GPU part = %f seconds\n", gpu_elapsed_time / 1000.0f); // Time is in ms
	/****************** The end of the GPU part: Do not modify anything in main() below this line  ************/
	
	//checking the correctness of the GPU part
	for(i = 0; i < n; i++)
	  if( fabs(temp[i] - c[i]) >= 0.009) //compare up to the second degit in floating point
		printf("Element %d in the result array does not match the sequential version\n", i);
		
	// Free the arrays in the host
	free(a); free(b); free(c); free(temp);
    
    // Destroy the events
    hipEventDestroy(gpu_start);
    hipEventDestroy(gpu_stop);

	return 0;
}


/**** TODO: Write the kernel itself below this line *****/
__global__ void vecGPU(float *ad, float *bd, float *cd, int n)
{
    // Calculate the global thread ID
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate the total number of threads in the grid
    int stride = gridDim.x * blockDim.x;

    // Use a grid-stride loop to process all elements
    for (int i = index; i < n; i += stride) {
        cd[i] += ad[i] * bd[i];
    }
}

